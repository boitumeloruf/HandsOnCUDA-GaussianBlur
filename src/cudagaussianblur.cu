#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//! Copyright 2017 Boitumelo Ruf. All rights reserved.
////////////////////////////////////////////////////////////////////////////////

#include "cudagaussianblur.cuh"

// std
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// opencv
#include <opencv2/imgproc.hpp>

// cuda runtime
#include <hip/hip_runtime.h>
#include <texture_types.h>

// cuda sdk
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples
#include <hip/hip_runtime_api.h>       // helper for checking cuda initialization and error checking
#include <helper_string.h>     // helper functions for string parsing

#include "cudautils_common.hpp"
#include "cudautils_devices.h"
#include "cudautils_memory.h"
#include "gaussianblur.kernel.cuh"


//==================================================================================================
cv::Mat runCudaGaussianBlur(const cv::Mat& inputImg)
{
  findCapableDevice();

  //--- adjust input image to Blocksize ---
  cv::Size imgSize = inputImg.size();
  imgSize.width -= ((imgSize.width % BLOCKSIZE_X) != 0) ?
                                 (imgSize.width % BLOCKSIZE_X) : 0;
  imgSize.height -= ((imgSize.height % BLOCKSIZE_Y) != 0) ?
                                (imgSize.height % BLOCKSIZE_Y) : 0;
  cv::resize(inputImg, inputImg, imgSize);


  //--- allocate memory ---
  //--- normalized texture coordinates are needed in order to use mirror border handler ---
  hipTextureObject_t inputImgTex = uploadImageToTexture<uint>(inputImg, hipAddressModeMirror, false);
  hipArray* outputSurf_data;
  hipSurfaceObject_t outputSurf = createSurfaceObject<uint>(imgSize, outputSurf_data);

  //--- run kernel ---
  dim3 numThreads = dim3(BLOCKSIZE_X, BLOCKSIZE_Y, 1);
  dim3 numBlocks = dim3(imgSize.width / numThreads.x,
                        imgSize.height/ numThreads.y);


  // First run the warmup kernel (which we'll use to get the GPU in the correct max power state
  applyGaussianBlur<<<numBlocks, numThreads>>>(inputImgTex, outputSurf, imgSize.width, imgSize.height);
  hipDeviceSynchronize();

  //--- Allocate CUDA events that we'll use for timing ---
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  printf("Launching CUDA Kernel\n");

  //--- Record the start event ---
  checkCudaErrors(hipEventRecord(start, NULL));

  //--- launch kernel ---
  applyGaussianBlur<<<numBlocks, numThreads>>>(inputImgTex, outputSurf, imgSize.width, imgSize.height);

  //--- Record the stop event ---
  checkCudaErrors(hipEventRecord(stop, NULL));

  //--- Wait for the stop event to complete ---
  checkCudaErrors(hipEventSynchronize(stop));

  //--- Check to make sure the kernel didn't fail ---
  getLastCudaError("Kernel execution failed");

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  //--- download result ---
  cv::Mat outputImg = downloadCudaArrayToImage<uint, uchar>(outputSurf_data, imgSize);

  printf("Input Size  [%dx%d], ", imgSize.width, imgSize.height);
  printf("GPU processing time : %.4f (ms)\n", msecTotal);
  printf("Pixel throughput    : %.3f Mpixels/sec\n",
         ((float)(imgSize.width * imgSize.height*1000.f)/msecTotal)/1000000);
  printf("------------------------------------------------------------------\n");

  // free memory
  checkCudaErrors(
        hipDestroyTextureObject(inputImgTex));
  checkCudaErrors(
        hipDestroySurfaceObject(outputSurf));

  return outputImg;
}
